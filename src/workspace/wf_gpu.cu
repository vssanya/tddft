#include "hip/hip_runtime.h"
#include "wf_gpu.h"

#include <hip/hip_runtime.h>
#include "../utils.h"


workspace::WfGpu::WfGpu(AtomCache const& atomCache, ShGrid const& grid, UabsCache const& uabsCache, int gpuGridNl, int threadsPerBlock):
	grid(grid),
	uabsCache(uabsCache),
	atomCache(atomCache),
	threadsPerBlock(threadsPerBlock)
{
    int Nr = grid.n[iR];

	d_gridNl = std::min(grid.n[iL], gpuGridNl);

    hipMalloc(&d_alpha, sizeof(cdouble)*Nr*d_gridNl);
    hipMalloc(&d_betta, sizeof(cdouble)*Nr*d_gridNl);

    hipMalloc(&d_uabs, sizeof(double)*Nr);
    hipMemcpy(d_uabs, uabsCache.data, sizeof(double)*Nr, hipMemcpyHostToDevice);

    hipMalloc(&d_atomU, sizeof(double)*Nr);
    hipMemcpy(d_atomU, atomCache.data_u, sizeof(double)*Nr, hipMemcpyHostToDevice);
}

workspace::WfGpu::~WfGpu() {
	hipFree(d_alpha);
	hipFree(d_betta);
	hipFree(d_uabs);
	hipFree(d_atomU);
}

__global__ void kernel_wf_prop_abs(hipComplex* wf, double* uabs, double dt, int Nr, int Nl) {
	int ir = blockIdx.x*blockDim.x + threadIdx.x;

	if (ir < Nr) {
		double u = exp(-uabs[ir]*dt);
		for (int il=0; il<Nl; il++) {
			wf[ir + il*Nr] *= u;
		}
	}
}

void workspace::WfGpu::prop_abs(ShWavefuncGPU& wf, double dt) {
	kernel_wf_prop_abs<<<div_up(grid.n[iR], threadsPerBlock), threadsPerBlock>>>((hipComplex*)wf.data, d_uabs, dt, grid.n[iR], grid.n[iL]);
}

// potentialType = 1 (POTENTIAL_COULOMB)
__global__ void kernel_wf_prop_at(hipComplex* wf, double* Ur, hipComplex* alpha, hipComplex* betta, int N, int Nr, int Nl, double dr, double dt, int Z, int potentialType) {
	int l0 = blockIdx.x*blockDim.x + threadIdx.x;

	double const dr2 = dr*dr;

	double const d2[3] = {1.0/dr2, -2.0/dr2, 1.0/dr2};
	double const d2_l0_11 = d2[1]*(1.0 - Z*dr/(12.0 - 10.0*Z*dr));

	double const M2[3] = {
		1.0/12.0,
		10.0/12.0,
		1.0/12.0
	};

	const double M2_l0_11 = (1.0 + d2_l0_11*dr2/12.0);

	double U[3];
	hipComplex al[3];
	hipComplex ar[3];
	hipComplex f;

	hipComplex* alpha_tid = &alpha[l0*Nr];
	hipComplex* betta_tid = &betta[l0*Nr];

	for (int l = l0; l < Nl; l+=N) {
		hipComplex* psi = &wf[l*Nr];

		auto Ul = [dr, Ur, l](int ir) -> double {
			double const r = dr*(ir+1);
			return l*(l+1)/(2*r*r) + Ur[ir];
		};

		hipComplex const idt_2 = 0.5*dt*hipComplex(0.0, 1.0);

		{
			int ir = 0;

			U[1] = Ul(ir  );
			U[2] = Ul(ir+1);

			for (int i = 1; i < 3; ++i) {
				al[i] = M2[i]*(1.0 + idt_2*U[i]) - 0.5*idt_2*d2[i];
				ar[i] = M2[i]*(1.0 - idt_2*U[i]) + 0.5*idt_2*d2[i];
			}

			if (l == 0 && potentialType == 1) {
				al[1] = M2_l0_11*(1.0 + idt_2*U[1]) - 0.5*idt_2*d2_l0_11;
				ar[1] = M2_l0_11*(1.0 - idt_2*U[1]) + 0.5*idt_2*d2_l0_11;
			}

			f = ar[1]*psi[ir] + ar[2]*psi[ir+1];

			alpha_tid[0] = -al[2]/al[1];
			betta_tid[0] = f/al[1];
		}

		for (int ir = 1; ir < Nr-1; ++ir) {
			U[0] = U[1];
			U[1] = U[2];
			U[2] = Ul(ir+1);

			for (int i = 0; i < 3; ++i) {
				al[i] = M2[i]*(1.0 + idt_2*U[i]) - 0.5*idt_2*d2[i];
				ar[i] = M2[i]*(1.0 - idt_2*U[i]) + 0.5*idt_2*d2[i];
			}

			hipComplex c = al[1] + al[0]*alpha_tid[ir-1];
			f = ar[0]*psi[ir-1] + ar[1]*psi[ir] + ar[2]*psi[ir+1];

			alpha_tid[ir] = - al[2] / c;
			betta_tid[ir] = (f - al[0]*betta_tid[ir-1]) / c;
		}

		{
			int ir = Nr-1;

			U[0] = U[1];
			U[1] = U[2];

			for (int i = 0; i < 2; ++i) {
				al[i] = M2[i]*(1.0 + idt_2*U[i]) - 0.5*idt_2*d2[i];
				ar[i] = M2[i]*(1.0 - idt_2*U[i]) + 0.5*idt_2*d2[i];
			}

			hipComplex c = al[1] + al[0]*alpha_tid[ir-1];
			f = ar[0]*psi[ir-1] + ar[1]*psi[ir];

			betta_tid[ir] = (f - al[0]*betta_tid[ir-1]) / c;
		}

		psi[Nr-1] = betta_tid[Nr-1];
		for (int ir = Nr-2; ir >= 0; --ir) {
			psi[ir] = alpha_tid[ir]*psi[ir+1] + betta_tid[ir];
		}
	}
}

void workspace::WfGpu::prop_at(ShWavefuncGPU& wf, double dt) {
	kernel_wf_prop_at<<<d_gridNl/threadsPerBlock, threadsPerBlock>>>((hipComplex*) wf.data, d_atomU,  (hipComplex*) d_alpha,  (hipComplex*) d_betta, d_gridNl, grid.n[iR], grid.n[iL], grid.d[iR], dt, atomCache.atom.Z, atomCache.atom.potentialType == Atom::PotentialType::POTENTIAL_COULOMB ? 1 : 0);
}

__device__ void cu_wf_dot(hipComplex v[2]) {
	hipComplex res[2] = {
		v[0] + v[1],
		-v[0] + v[1]
	};

	v[0] = res[0];
	v[1] = res[1];
}

__device__ void cu_wf_dot_T(hipComplex v[2]) {
	hipComplex res[2] = {
		0.5*(v[0] - v[1]),
		0.5*(v[0] + v[1])
	};

	v[0] = res[0];
	v[1] = res[1];
}

__global__ void kernel_wf_prop_ang_l(hipComplex* wf, hipComplex dt, const hipComplex eigenval0, const hipComplex eigenval1, int m, int l, int l1, double E, double dr, int Nr, int Nl) {
	int ir = blockIdx.x*blockDim.x + threadIdx.x;

	hipComplex* psi_l0 = &wf[Nr*l];
	hipComplex* psi_l1 = &wf[Nr*(l+l1)];

	hipComplex i = hipComplex(0.0, 1.0);

	if (ir < Nr) {
		double const r = dr*(ir+1);
		double const U = r*E*clm(l, m);

		hipComplex x[2] = {psi_l0[ir], psi_l1[ir]};

		cu_wf_dot(x);
		x[0] *= exp(i*U*dt*eigenval0);
		x[1] *= exp(i*U*dt*eigenval1);
		cu_wf_dot_T(x);

		psi_l0[ir] = x[0];
		psi_l1[ir] = x[1];
	}
}

void workspace::WfGpu::prop(ShWavefuncGPU& wf, field_t const& field, double t, double dt) {
	double E = field_E(&field, t + dt/2);

	dim3 gridDim(div_up(grid.n[iR], threadsPerBlock));

	const int l_max = 2;
	const int Nl = grid.n[iL];

	cdouble eigenval0 = {-1.0, 0.0};
	cdouble eigenval1 = {1.0, 0.0};

	for (int l1 = 1; l1 < l_max; ++l1) {
		for (int il = 0; il < Nl - l1; ++il) {
			kernel_wf_prop_ang_l<<<gridDim, threadsPerBlock>>>((hipComplex*) wf.data, 0.5*dt, eigenval0, eigenval1, wf.m, il, l1, E, grid.d[iR], grid.n[iR], Nl);
		}
	}

	prop_at(wf, dt);

	for (int l1 = l_max-1; l1 > 0; --l1) {
		for (int il = Nl - 1 - l1; il >= 0; --il) {
			kernel_wf_prop_ang_l<<<gridDim, threadsPerBlock>>>((hipComplex*) wf.data, 0.5*dt, eigenval0, eigenval1, wf.m, il, l1, E, grid.d[iR], grid.n[iR], Nl);
		}
	}

	prop_abs(wf, dt);
}
